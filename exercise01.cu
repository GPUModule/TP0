#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include "hip/hip_runtime.h"
#include ""
#include "utils.h"

#define N 16 //width
#define M 16 //height
#define THREADS_PER_BLOCK 256
#define SQRT_THREADS_PER_BLOCK sqrt(THREADS_PER_BLOCK)

void checkCUDAError(const char*);
void random_floats(float *a);
void MatMulCPU(float *a, float *b, float *c);
int validate(float *a, float *ref);


__global__ void MatMulKernel(float *a, float *b, float *c) {
	// A completer
}



int main(void) {
	srand( time( NULL ) );

	float *a, *b, *c, *c_ref;		// host copies of a, b, c
	float *d_a, *d_b, *d_c;			// device copies of a, b, c
	int errors;
	unsigned int size = N * M * sizeof(float);

	// Alloc space for device copies of a, b, c
	// A completer
	//hipMalloc
	//hipMalloc
	//hipMalloc
	checkCUDAError("CUDA malloc");

	// Alloc space for host copies of a, b, c and setup input values
	// A completer
	// a = 
	// b =
	// c =
	// c_ref =

	// Copy inputs to device
	// A completer
	//hipMemcpy
	//hipMemcpy
	checkCUDAError("CUDA memcpy");

	// Launch add() kernel on GPU
	//unsigned int block_width = 
	//unsigned int grid_width = 
	//unsigned int grid_height = 
	//dim3 blocksPerGrid(grid_width, grid_height);
	//dim3 threadsPerBlock(block_width, block_width);
	MatMulKernel << <blocksPerGrid, threadsPerBlock >> >(d_a, d_b, d_c);
	checkCUDAError("CUDA kernel");

	//perform CPU version
	MatMulCPU(a, b, c_ref);


	// Copy result back to host
	// A completer
	// hipMemcpy
	checkCUDAError("CUDA memcpy");

	// validate
	errors = validate(c, c_ref);
	printf("CUDA GPU result has %d errors.\n", errors);



	// Cleanup
	free(a); free(b); free(c);
	hipFree(d_a); hipFree(d_b); hipFree(d_c);
	checkCUDAError("CUDA cleanup");

	return 0;
}

void checkCUDAError(const char *msg)
{
	hipError_t err = hipGetLastError();
	if (hipSuccess != err)
	{
		fprintf(stderr, "CUDA ERROR: %s: %s.\n", msg, hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
}

void random_floats(float *a)
{
	for (unsigned int x = 0; x < N; x++){
		for (unsigned int y = 0; y < M; y++){
			a[(y*N) + x] = (float)(rand() % 101);
		}
	}
}

void MatMulCPU(float *a, float *b, float *c)
{
	// A completer
}

int validate(float *a, float *ref){
	int errors = 0;
	for (unsigned int x = 0; x < N; x++){
		for (unsigned int y = 0; y < M; y++){
			int i = (y*N) + x;
			if (a[i] != ref[i]){
				errors++;
				fprintf(stderr, "ERROR at index %d: GPU result %f does not match CPU value of %f\n", i, a[i], ref[i]);
			}
		}
	}

	return errors;
}